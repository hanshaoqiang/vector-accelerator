#include "hip/hip_runtime.h"
﻿extern "C" __global__ 
    void ZipExample(float** a, float** b, float* c, float** result, int n)
    {
        size_t tid = blockIdx.x * blockDim.x + threadIdx.x; 
        if (tid < n) 
        {
            float local0 = 0;
			for (int i = 0; i < n; ++i)
			{
				local0 = a[i][tid] * b[i][tid] * c[i];
			}
			result[tid] = local0;
        }
    }

	